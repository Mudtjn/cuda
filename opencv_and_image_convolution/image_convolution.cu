#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#define THREADS_PER_BLOCK 1024
#define KERNEL_SIZE 31

__global__
void lightenImage(float *d_image, float *d_result_image, int N) {
    // indx being pointed to
    int idx = (blockDim.x * blockIdx.x + threadIdx.x) ; 
    if(idx < N) {
        d_result_image[idx] = d_image[idx] * 3.0;
    }
}

void displayImage( cv::Mat &result_image) {
    cv::Mat display; 
    result_image.convertTo(display, CV_8UC3, 255.0); 
    cv::imshow("Output", display);
    cv::waitKey(0);
}

int main() {

    cv::Mat image = cv::imread("mario.png", cv::IMREAD_COLOR); 
    if(image.empty()) {
        std::cerr << "Couldn't load sample \n"; 
        return -1; 
    }

    // this is a 1-D array
    cv::Mat imgFloat; 
    image.convertTo(imgFloat, CV_32FC3, 1.0/255.0); 

    std::cout << "Image size: " << imgFloat.rows << " x " << imgFloat.cols << std::endl;

    float *initial_image, *result_image;
    size_t num_elements = imgFloat.total() * imgFloat.channels(); 
    size_t image_size = num_elements * sizeof(float); 
    


    hipMallocManaged(&initial_image, image_size); 
    hipMallocManaged(&result_image, image_size); 
    hipMemcpy(initial_image, imgFloat.ptr(), image_size, hipMemcpyHostToDevice); 

    auto BLOCKS_PER_GRID = (num_elements + THREADS_PER_BLOCK - 1) /   THREADS_PER_BLOCK; 

    lightenImage<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>> (initial_image, result_image, image_size);
    hipDeviceSynchronize(); 
    
    cv::Mat final_image(imgFloat.rows, imgFloat.cols, CV_32FC3, result_image); 
    displayImage(final_image); 

    return 0; 
}