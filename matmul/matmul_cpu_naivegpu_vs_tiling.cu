#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
using namespace std; 

#define M 4096  // Number of rows in A and C
#define K 8192   // Number of columns in A and rows in B
#define N 4096  // Number of columns in B and C
// keeping tile = warp size for better coalescing
#define BLOCK_SIZE 32

#define CHECK_CUDA(call) do { \
    hipError_t _err = (call); \
    if (_err != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(_err)); \
        exit(1); \
    } \
} while(0)


void matmul_cpu(float* A, float* B, float *C, int m, int k, int n) {
    for(int i =0 ; i<m ; i++) {
        for(int j = 0; j<n ; j++) {
            float sum = 0.0f; 
            for(int p = 0 ; p<k ; p++) {
                sum += A[i*k + p] * B[p*n + j];
            }
            C[i*n + j] = sum;
        }
    }
}

__global__ 
void matmul_gpu(float* A, float* B, float * C, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n) {
        float sum = 0.0f ;
        for(int i = 0 ; i<k ; i++) {
            sum += A[row * k + i] * B[i * n + col]; 
        }
        C[row * n + col] = sum;
    }
}

__global__
void matmul_gpu_tiling(const float* A, const float* B, float* C, int m, int k, int n) {
    __device__ __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __device__ __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Conventional mapping: threadIdx.x -> column within tile, threadIdx.y -> row within tile
    int tileRow = blockIdx.y;      // which tile-row of C
    int tileCol = blockIdx.x;      // which tile-col of C
    int localRow = threadIdx.y;    // 0..BLOCK_SIZE-1
    int localCol = threadIdx.x;    // 0..BLOCK_SIZE-1
 
    unsigned int row = tileRow * BLOCK_SIZE + localRow; // global row index in C
    unsigned int col = tileCol * BLOCK_SIZE + localCol; // global col index in C

    float sum = 0.0f; // this is each thread memory 

    int numTiles = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for (int t = 0; t < numTiles; ++t) {
        // Load tile of A: rows = row, cols = t*BLOCK_SIZE + localCol
        int aCol = t * BLOCK_SIZE + localCol;
        if (row < m && aCol < k)
            As[localRow][localCol] = A[row * k + aCol];
        else
            As[localRow][localCol] = 0.0f;

        // Load tile of B: rows = t*BLOCK_SIZE + localRow, cols = col
        int bRow = t * BLOCK_SIZE + localRow;
        if (bRow < k && col < n)
            Bs[localRow][localCol] = B[bRow * n + col];
        else
            Bs[localRow][localCol] = 0.0f;

        __syncthreads();

        // Multiply accumulate for this tile
        for (int e = 0; e < BLOCK_SIZE; ++e) {
            sum = __fadd_rd(sum, __fmul_rd(As[localRow][e], Bs[e][localCol])); 
        }
        __syncthreads();
    }

    if (row < m && col < n) {
        C[row * n + col] = sum; // write once after accumulating all tiles
    }
}


// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {

    float *h_A, *h_B, *h_C_cpu;
    float *d_A, *d_B, *d_C, *d_C_tiling;
    
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);

    srand(time(NULL)); 
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));
    CHECK_CUDA(hipMalloc(&d_C_tiling, size_C));

    CHECK_CUDA(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE); // 1024 threads per block
    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE); // grid => 8 * 8
    // 64 thread blocks
    // 256 * 256 / 8*8 = 1024 operations per block
    // 1024 / 1024 = 1 operation per thread

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        // matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        matmul_gpu<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, K, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipDeviceSynchronize());
        matmul_gpu_tiling<<<gridSize, blockSize>>>(d_A, d_B, d_C_tiling, M, K, N); 
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // Benchmark CPU implementation
    // printf("Benchmarking CPU implementation...\n");
    // double cpu_total_time = 0.0;
    // for (int i = 0; i < 20; i++) {
    //     double start_time = get_time();
    //     matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
    //     double end_time = get_time();
    //     cpu_total_time += end_time - start_time;
    // }
    // double cpu_avg_time = cpu_total_time / 20.0;

    // Benchmark GPU implementation
    printf("Benchmarking GPU implementation...\n");
    /**
     * @brief This is cuda time recording 
     * measuring with get_time also includes 
     * kernel launch overhead
     * driver scheduling
     * OS jitter
     * PCIe synchronization costs
     * 
     * This will cause computed time for gpu to be all over the place as other are variable
     * hipEventCreate help calculate raw cost of computation
     */
    hipEvent_t start, stop; 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); 
    for(int i = 0 ; i<20 ; i++) {    
        matmul_gpu<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, K, N);
    }
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    float matmul_naive = 0.0f; 
    hipEventElapsedTime(&matmul_naive, start, stop);
    double gpu_avg_time = matmul_naive/ 20.0;

    // Benchmarking GPU tiling
    printf("Benchmarking GPU TILING implementation...\n");
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); 
    for(int i = 0 ; i<20 ; i++) {
        matmul_gpu_tiling<<<gridSize, blockSize>>>(d_A, d_B, d_C_tiling, M, K, N);
    }
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    float matmul_tiling = 0.0f; 
    hipEventElapsedTime(&matmul_tiling, start, stop); 
    double gpu_avg_time_tiling = matmul_tiling / 20.0;


    hipEventDestroy(start); 
    hipEventDestroy(stop); 

    // Print results
    // printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e3f));
    printf("GPU average time tiling: %f microseconds\n", (gpu_avg_time_tiling * 1e3f));
    // printf("Speedup: %fx\n", cpu_avg_time * 1e3f / gpu_avg_time);
    printf("Speed gpu vs gpu_tiling: %fx\n", gpu_avg_time / gpu_avg_time_tiling); 

    free(h_A); 
    free(h_B);
    free(h_C_cpu);
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));  
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_C_tiling));

    return 0; 
}

// Performing warm-up runs...
// Benchmarking GPU implementation...
// Benchmarking GPU TILING implementation...
// GPU average time: 408633.544922 microseconds
// GPU average time tiling: 281615.869141 microseconds
// Speed gpu vs gpu_tiling: 1.451032x